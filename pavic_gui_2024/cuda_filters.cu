
#include <hip/hip_runtime.h>
__global__ void apply_bw_filter_cuda(unsigned char* d_pixelData, int width, int height, int intensity) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (x + y * width) * 3;

        int red = d_pixelData[idx];
        int green = d_pixelData[idx + 1];
        int blue = d_pixelData[idx + 2];

        // Calculate grayscale value
        int grayValue = (int)(0.299 * red + 0.587 * green + 0.114 * blue);

        // Adjust the colors based on the intensity
        red = red + ((grayValue - red) * intensity) / 100;
        green = green + ((grayValue - green) * intensity) / 100;
        blue = blue + ((grayValue - blue) * intensity) / 100;

        // Write the new RGB values back
        d_pixelData[idx] = red;
        d_pixelData[idx + 1] = green;
        d_pixelData[idx + 2] = blue;
    }
}